#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "cuda_intrinsics.h" 

#define FLTSIZE sizeof(float)

class WarpTrace
{
public:
	WarpTrace() {
		trace_sm 		= NULL;
		trace_blk 		= NULL;
		trace_start   	= NULL;
		trace_end     	= NULL;
		total_warps   	= 0;
		total_threads 	= 0;
	}

	~WarpTrace() {
		free_data();	
	}

	void compute_totalwarps(dim3 blocks, dim3 threads)
	{
		int total_blocks = blocks.x * blocks.y * blocks.z;	

		int warps_per_block = int(ceil(threads.x / 32.f) * 
				ceil(threads.y / 32.f) * ceil(threads.z / 32.f));

		total_warps = total_blocks * warps_per_block;
	}

	void compute_totalthreads(dim3 blocks, dim3 threads)
	{
		int total_blocks = blocks.x * blocks.y * blocks.z;	
		int threads_per_blk = threads.x * threads.y * threads.z; 
		total_threads= total_blocks * threads_per_blk;
	}

	void print_log(int streamid)
	{
		hipDeviceSynchronize();

		for(int i=0; i<total_threads; i++)
		{
			printf("%12d,%12d,%12u,%12u,%12lf,%12lf\n",
					i, 
					streamid, 
					trace_sm[i], 
					trace_blk[i],
					trace_start[i],
					trace_end[i]);
		}
	}

	void allocate_data() 
	{
		hipMallocManaged((void**)&trace_sm,    total_threads * sizeof(uint));
		hipMallocManaged((void**)&trace_blk,   total_threads * sizeof(uint));
		hipMallocManaged((void**)&trace_start, total_threads * sizeof(double));
		hipMallocManaged((void**)&trace_end,   total_threads * sizeof(double));
	}

	void free_data()
	{
		if(trace_sm    != NULL)	hipFree(trace_sm);
		if(trace_blk   != NULL)	hipFree(trace_blk);
		if(trace_start != NULL)	hipFree(trace_start);
		if(trace_end   != NULL)	hipFree(trace_end);
	}

	uint* trace_sm;
	uint* trace_blk;
	double* trace_start;
	double* trace_end;

	uint total_warps;
	uint total_threads;
};


inline int BLK(int data, int blocksize)
{
	return (data + blocksize - 1) / blocksize;
}


__global__ void kernel_vectorAdd (const float* __restrict__ a_d, 
		const float* __restrict__ b_d,
		const int N,
		const int offset,
		float *c_d,
		uint *trace_sm,
		uint *trace_blk,
		double *trace_start,
		double *trace_end
		)
{
	int tid = threadIdx.x + __mul24(blockIdx.x, blockDim.x);

	trace_sm[tid] 	= get_smid();
	trace_blk[tid]	= get_global_blkid(); 

	//-----------------------------//
	// start timing 
	//-----------------------------//
	double start_time = double(timer()) * 1e-6; 
	trace_start[tid] = start_time;

	//-----------------------------//
	// program
	//-----------------------------//
	if(tid < N) {
		// read the next 16 data
		float sum = 0.f;
		for(int i=0; i<128; i++) {
			if(tid + i < N) {
				sum += a_d[tid + i + offset] + b_d[tid + i + offset];	
			}	
		}
		c_d[tid + offset] = sum; 
	}

	//-----------------------------//
	// finish timing 
	//-----------------------------//
	double end_time = double(timer()) * 1e-6;
	trace_end[tid] = end_time;
}


int main( int argc, char **argv)
{
	int devid = 0 ;

	int num_streams = 8;

	int N = 1 << 20;

	int dump_trace = 0;

	if(argc >= 2)
		num_streams = atoi(argv[1]);

	if(argc >= 3)
		devid = atoi(argv[2]);

	if(argc >= 4)
		N = atoi(argv[3]);

	if(argc >= 5)
		dump_trace = atoi(argv[4]);


	hipSetDevice(devid);

	// allocate streams
    hipStream_t *streams = (hipStream_t *) malloc(num_streams * sizeof(hipStream_t));

	// init
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamCreate(&(streams[i])));
    }

	//------------------------------------------------------------------------//
	// allocate data on the host
	//------------------------------------------------------------------------//
	size_t databytes = N  * FLTSIZE; 

	float *a_h = NULL;
    checkCudaErrors(hipHostMalloc((void **)&a_h, N * num_streams * FLTSIZE));

	float *b_h = NULL;
    checkCudaErrors(hipHostMalloc((void **)&b_h, N * num_streams * FLTSIZE));

	float *c_h = NULL;
    checkCudaErrors(hipHostMalloc((void **)&c_h, N * num_streams * FLTSIZE));

	for(int i=0; i< N * num_streams; i++) {
		a_h[i] = 1.1f;	
		b_h[i] = 2.2f;	
	}

	//------------------------------------------------------------------------//
	// allocate data on the device 
	//------------------------------------------------------------------------//
	float *a_d;
	float *b_d;
	float *c_d;
	hipMalloc((void**)&a_d, N * num_streams * FLTSIZE);
	hipMalloc((void**)&b_d, N * num_streams * FLTSIZE);
	hipMalloc((void**)&c_d, N * num_streams * FLTSIZE);

	// kernel configuration
	dim3 threads = dim3(256, 1, 1);
	dim3 blocks  = dim3(BLK(N, threads.x), 1, 1);

	WarpTrace *streams_trace = new WarpTrace[num_streams];

	for(int i=0; i<num_streams; i++) {
		streams_trace[i].compute_totalthreads(blocks, threads);
		streams_trace[i].allocate_data();
	}

	// create cuda event handles
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	hipEventRecord(start,0);

	// copy data to deivce
	for (int i = 0; i < num_streams; i++) {
		int offset = i * N;
		hipMemcpyAsync(&a_d[offset], &a_h[offset],  databytes, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(&b_d[offset], &b_h[offset],  databytes, hipMemcpyHostToDevice, streams[i]);
	}

	// launch one worker kernel per stream
	for (int i = 0; i < num_streams; i++) {
		int offset = i * N;
		kernel_vectorAdd <<< blocks, threads, 0, streams[i] >>> (a_d, 
				                                                 b_d, 
																 N, 
																 offset,
																 c_d,
																 streams_trace[i].trace_sm,
																 streams_trace[i].trace_blk,
																 streams_trace[i].trace_start,
																 streams_trace[i].trace_end
																 );
	}

	// copy data back to host
	for (int i = 0; i < num_streams; i++) {
		int offset = i * N;
		hipMemcpyAsync(&c_h[offset], &c_d[offset],  databytes, hipMemcpyDeviceToHost, streams[i]);
	}

	// required for async copy
	hipDeviceSynchronize();

    hipEventRecord(stop, 0);
	
	float gpuTime_ms= 0;
	hipEventElapsedTime(&gpuTime_ms, start, stop);

	//printf("runtime (ms) : %f\n", gpuTime_ms);

	if(dump_trace == 1) {

		printf("thread_id,stream_id,sm_id,block_id,start,end\n");

		for (int i = 0; i < num_streams; i++) {
			streams_trace[i].print_log(i);
		}
	}

	//------------------------------------------------------------------------//
	// free 
	//------------------------------------------------------------------------//
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamDestroy(streams[i]));
    }

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

	hipHostFree(a_h);
	hipHostFree(b_h);
	hipHostFree(c_h);

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	delete [] streams_trace;

	hipDeviceReset();

	return 0;
}
